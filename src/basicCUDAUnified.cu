#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define tile_width 32

__global__ void gpu_matrix_mult(double *a, double *b, double *c, int n)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col < n && row < n)
  {
    double sum = 0;
    for (int i = 0; i < n; i++)
    {
      sum += a[row * n + i] * b[i * n + col];
    }
    c[row * n + col] = sum;
  }

}


int main(){

    int block_size = 128, matrix_size = 1024;

    //Create and allocate memory for the matrices
    double *a, *b, *c;
    hipMallocManaged((void **)&a, sizeof(double) * matrix_size * matrix_size);
    hipMallocManaged((void **)&b, sizeof(double) * matrix_size * matrix_size);
    hipMallocManaged((void **)&c, sizeof(double) * matrix_size * matrix_size);

    // initialize matrix A
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        a[i * matrix_size + j] = (double(rand() % 50))/10.0;
      }
    }

    // initialize matrix B
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        b[i * matrix_size + j] = (double(rand() % 50))/10.0;
      }
    }

    // initialize matrix C
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        c[i * matrix_size + j] = 0;
      }
    }

    float gpu_elapsed_time_ms;

    // create cuda events for timing purposes
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);

    //Define block size
    unsigned int grid_rows = (matrix_size + block_size - 1) / block_size;
    unsigned int grid_cols = (matrix_size + block_size - 1) / block_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);

    //Launch GPU Kernel
    hipEventRecord(gpu_start, 0);
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(a, b, c, matrix_size);
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);

    //Calculate elapsed time
    hipEventElapsedTime(&gpu_elapsed_time_ms, gpu_start, gpu_stop);
    printf("Time elapsed on Naive matrix multiplication of %dx%d on GPU: %f ms.\n\n",matrix_size, matrix_size, gpu_elapsed_time_ms);

    // free memory 
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}