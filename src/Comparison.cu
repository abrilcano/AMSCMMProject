#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define tile_width 32

__global__ void tiled_matrix_mult(double *a,double *b, double *c, int n)
{
    __shared__ double ds_M[tile_width][tile_width];
    __shared__ double ds_N[tile_width][tile_width];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * blockDim.y + ty;
  int Col = bx * blockDim.x + tx;
  double Pvalue = 0;

  // Loop over the M and N tiles required to compute the P element
  for (int p = 0; p < (n-1) / tile_width + 1; ++p) {
    // Collaborative loading of M and N tiles into shared memory
    if(Row < n && p * tile_width+tx < n) {
        ds_M[ty][tx] = a[Row*n + p*tile_width+tx];
    }
    else
    {
        ds_M[ty][tx] = 0.0;
    }
    if (p*tile_width+ty < n && Col < n) {
        ds_N[ty][tx] = b[(p*tile_width+ty)*n + Col];
    }
    else
    {
        ds_N[ty][tx] = 0.0;
    }
    __syncthreads();

    if(Row < n && Col < n) {
        for (int i = 0; i < tile_width; ++i)
           Pvalue += ds_M[ty][i] * ds_N[i][tx];
    }
    __syncthreads();
  }
  if (Row < n && Col < n)
    c[Row*n+Col] = Pvalue;
}

__global__ void gpu_matrix_mult(double *a, double *b, double *c, int n)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col < n && row < n)
  {
    int sum = 0;
    for (int i = 0; i < n; i++)
    {
      sum += a[row * n + i] * b[i * n + col];
    }
    c[row * n + col] = sum;
  }
  
}


int main(){

    int block_size = 256, matrix_size = 8192;

    //Create and allocate memory for the matrices
    double *a, *b, *cn, *ct;
    hipMallocManaged((void **)&a, sizeof(double) * matrix_size * matrix_size);
    hipMallocManaged((void **)&b, sizeof(double) * matrix_size * matrix_size);
    hipMallocManaged((void **)&cn, sizeof(double) * matrix_size * matrix_size);
    hipMallocManaged((void **)&ct, sizeof(double) * matrix_size * matrix_size);

    // initialize matrix A
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        a[i * matrix_size + j] = (double(rand() % 50))/10.0;
      }
    }

    // initialize matrix B
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        b[i * matrix_size + j] = (double(rand() % 50))/10.0;
      }
    }

    // initialize matrix C
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        cn[i * matrix_size + j] = 0;
      }
    }

    // initialize matrix C
    for (int i = 0; i < matrix_size; ++i){
      for (int j = 0; j < matrix_size; ++j){
        ct[i * matrix_size + j] = 0;
      }
    }

    float tiled_time;
    float naive_time;

    // create cuda events for timing purposes
    hipEvent_t gpu_startNaive, gpu_stopNaive;
    hipEventCreate(&gpu_startNaive);
    hipEventCreate(&gpu_stopNaive);
    hipEvent_t gpu_startTiled, gpu_stopTiled;
    hipEventCreate(&gpu_startTiled);
    hipEventCreate(&gpu_stopTiled);

    //Define block size
    unsigned int grid_rows = (matrix_size + block_size - 1) / block_size;
    unsigned int grid_cols = (matrix_size + block_size - 1) / block_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);

    //Launch GPU Kernel
    hipEventRecord(gpu_startNaive, 0);
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(a, b, cn, matrix_size);
    hipEventRecord(gpu_stopNaive, 0);
    hipEventSynchronize(gpu_stopNaive);

    //Calculate elapsed time
    hipEventElapsedTime(&naive_time, gpu_startNaive, gpu_stopNaive);
    printf("Time elapsed on Naive matrix multiplication of %dx%d on GPU: %f ms.\n\n",matrix_size, matrix_size, naive_time);
           
    //Launch GPU Kernel
    hipEventRecord(gpu_startTiled, 0);
    tiled_matrix_mult<<<dimGrid, dimBlock>>>(a, b, ct, matrix_size);
    hipEventRecord(gpu_stopTiled, 0);
    hipEventSynchronize(gpu_stopTiled);

    //Calculate elapsed time
    hipEventElapsedTime(&tiled_time, gpu_startTiled, gpu_stopTiled);
    printf("Time elapsed on Tiled matrix multiplication of %dx%d on GPU: %f ms.\n\n",matrix_size, matrix_size, tiled_time);

    // free memory 
    hipFree(a);
    hipFree(b);
    hipFree(cn);
    hipFree(ct);

    return 0;
}

